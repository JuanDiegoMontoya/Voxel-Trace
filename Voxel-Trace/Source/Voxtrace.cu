#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "Voxtrace.h"
#include "RayCamera.h"

#include "Renderer.h"
#include <Engine.h>
#include <Pipeline.h>
#include <camera.h>
#include <Line.h>
#include <shader.h>

#include <vbo.h>
#include <vao.h>

#include "CommonDevice.cuh"
#include "cuda_gl_interop.h"
#include "pick.h"

surface<void, 2> screenSurface;

__device__ static bool swagCallback(glm::vec3 p, Voxels::Block* block, glm::vec3 norm)
{
	if (block)
	{
		//printf("hit pos: %.0f, %.0f, %.0f\n", p.x, p.y, p.z);
		return true;
	}
	return false;
}

__global__ static void epicRayTracer(Voxels::Block* pWorld, glm::ivec3 worldDim,
	PerspectiveRayCamera cam, 
	glm::vec3 chunkDim, glm::vec2 imgSize, float time)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int n = imgSize.x * imgSize.y;

	//printf("index = %d, stride = %d, n = %d\n", index, stride, n);
	for (int i = index; i < n; i += stride)
	{
		glm::vec2 imgPos = expand(i, imgSize.y);
		//glm::vec2 imgPos(x, y);
		glm::vec2 screenCoord(
			(2.0f * imgPos.x) / imgSize.x - 1.0f,
			(-2.0f * imgPos.y) / imgSize.y + 1.0f);
		Ray ray = cam.makeRay(screenCoord);
		glm::vec4 val = glm::vec4(ray.direction * .5f + glm::vec3(1), 1);

		val = { .53f, .81f, .92f, 1 };


		auto cb = [&pWorld, &worldDim, &val, &ray](glm::vec3 p, Voxels::Block* block, glm::vec3 norm, glm::vec3 ex)->bool
		{
			if (block)
			{
				if (block->alpha == 0)
					return false;
				//printf("hit pos: %.0f, %.0f, %.0f\n", p.x, p.y, p.z);
				glm::vec3 FragColor(0);
				//FragColor = block->diffuse;
				//FragColor(norm + glm::vec3(1)) * .5f;

				bool shadowed = false;
				auto shadowCB = [&shadowed](glm::vec3 p, Voxels::Block* block, glm::vec3 norm, glm::vec3)->bool
				{
					if (block && block->alpha == 1)
					{
						shadowed = true;
						return true;
					}
					return false;
				};
				glm::vec3 sunPos(20, 0, 0);
				glm::vec3 sunRay = glm::normalize(ex - sunPos); // block-to-light ray

				//glm::vec3 shadowDir = glm::reflect(-ray.direction, norm);
				glm::vec3 shadowDir(sunRay);
				raycastBranchless(pWorld, worldDim, ex + .1f * shadowDir, shadowDir, 100, shadowCB);
				block->diffuse = shadowDir * .5f + .5f;

				float diff = glm::max(glm::dot(sunRay, norm), 0.f);
				float spec = glm::pow(glm::max(glm::dot(ray.direction, glm::reflect(sunRay, norm)), 0.0f), 64.f);
				glm::vec3 ambient = glm::vec3(.2) * block->diffuse;
				glm::vec3 specular = glm::vec3(.7) * spec;
				glm::vec3 diffuse = block->diffuse * diff;
				if (shadowed)
				{
					diffuse = specular = { 0, 0, 0 };
				}
				FragColor = diffuse + ambient + specular;
				val = glm::vec4(FragColor, 1.f);
				return true;
			}
			return false;
		};

		raycastBranchless(pWorld, worldDim, ray.origin, ray.direction, 150.f, cb);

		// write final pixel value
		surf2Dwrite(val, screenSurface, imgPos.x * sizeof(val), imgSize.y - imgPos.y - 1);
	}
}

namespace Voxels
{
	namespace
	{
		PerspectiveRayCamera cam;
		LinePool* lines = nullptr;

		// world description
		Block* blocks = nullptr;
		glm::ivec3 chunkDim = { 10, 10, 10 };
		int numBlocks = chunkDim.x * chunkDim.y * chunkDim.z;

		// screen info
		glm::vec2 screenDim = { 500, 265 };
		//glm::vec2 screenDim = { 1920, 1080 }; // 1080p
		//glm::vec2 screenDim = { 1280, 720 };  // 720p
		//glm::vec2 screenDim = { 853, 480 };   // 480p
		//glm::vec2 screenDim = { 125, 65 };
		
		// rendering shiz
		VBO* vbo = nullptr;
		VAO* vao = nullptr;
		GLuint screenTexture = -1;

		// cuda GL stuff
		hipGraphicsResource* imageResource = nullptr;
		hipArray* arr = nullptr;

		const int KernelBlockSize = 256;
		const int KernelNumBlocks = (screenDim.x * screenDim.y + KernelBlockSize - 1) / KernelBlockSize;
	}

	void Init()
	{
		Engine::PushRenderCallback(Render, 4);
		InitGLStuff();
		InitBlocks();
	}

	void InitGLStuff()
	{
		// TODO: move this to Vertices.h or something
		float quadVertices[] =
		{
			// positions        // texture Coords
			-1.0f,  1.0f, 0.0f, 0.0f, 1.0f,
			-1.0f, -1.0f, 0.0f, 0.0f, 0.0f,
			 1.0f,  1.0f, 0.0f, 1.0f, 1.0f,
			 1.0f, -1.0f, 0.0f, 1.0f, 0.0f,
		};

		// setup screen texture pointers
		vbo = new VBO(&quadVertices[0],
			sizeof(quadVertices), GL_STATIC_DRAW);
		VBOlayout layout;
		layout.Push<float>(3); // pos
		layout.Push<float>(2); // texcoord
		vao = new VAO();
		vao->AddBuffer(*vbo, layout);

		// generate screen texture memory
		glGenTextures(1, &screenTexture);
		glBindTexture(GL_TEXTURE_2D, screenTexture);
		// cuda behavior becomes extremely weird when using RGB textures with it
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, 
			screenDim.x, screenDim.y, 0, GL_RGBA, GL_FLOAT, NULL);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
		glBindTexture(GL_TEXTURE_2D, 0);

		// init texture color
		glm::vec4 defColor(1, .1, .1, 1.);
		glClearTexImage(screenTexture, 0, GL_RGBA, GL_FLOAT, &defColor[0]);

		// register the texture as a cuda resource
		cudaCheck(
			hipGraphicsGLRegisterImage(&imageResource, screenTexture,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
	}

	void InitBlocks()
	{
		// shared memory so that the GPU and CPU can both read and write blocks
		cudaCheck(hipMallocManaged(&blocks, numBlocks * sizeof(Block)));

		for (int i = 0; i < numBlocks; i++)
		{
			blocks[i].alpha = 0;
			auto pos = expand(i, chunkDim.x, chunkDim.y);
			//if (glm::all(glm::lessThan(pos, { 5, 5, 5 })))
			{
				blocks[i].alpha = rand() % 100 > 50 ? 1 : 0;
				//blocks[i].diffuse = { 1, 0, 0 };
			}
			blocks[i].diffuse = Utils::get_random_vec3_r(0, 1);
		}
	}

	void Render()
	{
		auto c = Renderer::GetPipeline()->GetCamera(0);
		cam = PerspectiveRayCamera(c->GetPos(), c->GetPos() + c->GetDir(), 
			glm::vec3(0, 1, 0), glm::radians(30.f), screenDim.x / screenDim.y);

		// ray trace her
		{
			cudaCheck(hipGraphicsMapResources(1, &imageResource, 0));
			cudaCheck(hipGraphicsSubResourceGetMappedArray(&arr, imageResource, 0, 0));
			cudaCheck(cudaBindSurfaceToArray(screenSurface, arr));

			//printf("screenDim = %f, %f\n", screenDim.x, screenDim.y);
			epicRayTracer<<<KernelNumBlocks, KernelBlockSize>>>(
				blocks, chunkDim, cam, chunkDim, screenDim, glfwGetTime());
			hipDeviceSynchronize();

			cudaCheck(hipGraphicsUnmapResources(1, &imageResource, 0));
		}

		// draw fullscreen quad
		ShaderPtr s = Shader::shaders["fullscreen"];
		s->Use();
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, screenTexture);
		s->setInt("tex", 0);
		vao->Bind();
		glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);
		vao->Unbind();
		s->Unuse();

		glClear(GL_DEPTH_BUFFER_BIT);
		if (lines)
		{
			ShaderPtr s = Shader::shaders["line"];
			s->Use();
			glm::mat4 model(1);
			glm::mat4 view = c->GetView();
			glm::mat4 proj = c->GetProj();
			s->setMat4("u_model", model);
			s->setMat4("u_view", view);
			s->setMat4("u_proj", proj);
			lines->Draw();
		}
	}

	void CameraRaySnapshot()
	{
		delete lines; // ok if null (e.g. first instance)

		std::vector<glm::vec3> poss, dirs, tClrs, bClrs;

		glm::vec2 imgSize(screenDim);
		for (int x = 0; x < imgSize.x; x++)
		{
			for (int y = 0; y < imgSize.y; y++)
			{
				glm::vec2 screenCoord(
					(2.0f * x) / imgSize.x - 1.0f,
					(-2.0f * y) / imgSize.y + 1.0f);
				Ray ray = cam.makeRay(screenCoord);
				poss.push_back(ray.origin);
				dirs.push_back(ray.direction);
			}
		}

		for (int i = 0; i < poss.size(); i++)
		{
			tClrs.push_back(glm::vec3(1));
			bClrs.push_back(glm::vec3(0));
		}

		lines = new LinePool(poss, dirs, tClrs, bClrs);
	}
}
