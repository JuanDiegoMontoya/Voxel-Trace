#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "Voxtrace.h"
#include "RayCamera.h"

#include "Renderer.h"
#include <Engine.h>
#include <Pipeline.h>
#include <camera.h>
#include <Line.h>
#include <shader.h>

#include <vbo.h>
#include <vao.h>

#include "CommonDevice.cuh"
#include "cuda_gl_interop.h"

surface<void, 2> screenSurface;

__global__ static void epicRayTracer(PerspectiveRayCamera cam, 
	glm::vec3 chunkDim, glm::vec2 imgSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int n = imgSize.x * imgSize.y;

	printf("index = %d, stride = %d, n = %d\n", index, stride, n);
	for (int i = index; i < n; i += stride)
	{
		glm::vec2 imgPos = expand(i, imgSize.x);
		glm::vec2 screenCoord(
			(2.0f * imgPos.x) / imgSize.x - 1.0f,
			(-2.0f * imgPos.y) / imgSize.y + 1.0f);
		//Ray ray = cam.makeRay(screenCoord);
		float3 val = { 1, 1, 1 };
		surf2Dwrite(val, screenSurface,
			imgPos.x * sizeof(float3), imgPos.y);
		//printf("i = %d, imgpos = %f, %f\n", i, imgPos.x, imgPos.y);
	}
}

namespace Voxels
{
	namespace
	{
		PerspectiveRayCamera cam;
		LinePool* lines = nullptr;

		// world description
		Block* blocks = nullptr;
		glm::vec3 chunkDim = { 10, 10, 10 };
		int numBlocks = chunkDim.x * chunkDim.y * chunkDim.z;

		// screen info
		glm::vec2 screenDim = { 20, 10 };
		
		// rendering shiz
		VBO* vbo = nullptr;
		VAO* vao = nullptr;
		GLuint screenTexture = -1;

		// cuda GL stuff
		hipGraphicsResource* imageResource;
		hipArray* arr;
	}

	void Init()
	{
		Engine::PushRenderCallback(Render, 5);
		InitGLStuff();
	}

	void InitGLStuff()
	{
		// TODO: move this to Vertices.h or something
		std::vector<glm::vec2> screenTexCoords =
		{
			{-1,-1 }, { 0, 0 },
			{ 1,-1 }, { 1, 0 },
			{ 1, 1 }, { 1, 1 },
			{-1,-1 }, { 0, 0 },
			{ 1, 1 }, { 1, 1 },
			{-1, 1 }, { 0, 1 },
		};

		// setup screen texture pointers
		vbo = new VBO(&screenTexCoords[0], 
			screenTexCoords.size() * sizeof(glm::vec2), GL_STATIC_DRAW);
		VBOlayout layout;
		layout.Push<float>(2); // pos
		layout.Push<float>(2); // texcoord
		vao = new VAO();
		vao->AddBuffer(*vbo, layout);

		// generate screen texture memory
		glGenTextures(1, &screenTexture);
		glBindTexture(GL_TEXTURE_2D, screenTexture);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, 
			screenDim.x, screenDim.y, 0, GL_RGB, GL_FLOAT, NULL);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
		glBindTexture(GL_TEXTURE_2D, 0);

		// init texture color
		glm::vec3 defColor(0, 1, 0); // cyan
		glClearTexImage(screenTexture, 0, GL_RGB, GL_FLOAT, &defColor[0]);

		// register the texture as a cuda resource
		cudaCheck(hipGraphicsGLRegisterImage(&imageResource, screenTexture,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
	}

	void Render()
	{
		auto c = Renderer::GetPipeline()->GetCamera(0);
		cam = PerspectiveRayCamera(c->GetPos(), c->GetPos() + c->GetDir(), 
			glm::vec3(0, 1, 0), glm::radians(30.f), 1920.f / 1080.f);
		
		if (lines)
		{
			ShaderPtr s = Shader::shaders["line"];
			s->Use();
			glm::mat4 model(1);
			glm::mat4 view = c->GetView();
			glm::mat4 proj = c->GetProj();
			s->setMat4("u_model", model);
			s->setMat4("u_view", view);
			s->setMat4("u_proj", proj);
			lines->Draw();
		}

		// ray trace her
		if (1)
		{
			cudaCheck(hipGraphicsMapResources(1, &imageResource, 0));
			cudaCheck(hipGraphicsSubResourceGetMappedArray(&arr, imageResource, 0, 0));
			cudaCheck(cudaBindSurfaceToArray(screenSurface, arr));

			printf("screenDim = %f, %f\n", 
				screenDim.x, screenDim.y);
			epicRayTracer<<<1, 1>>>(cam, chunkDim, screenDim);
			hipDeviceSynchronize();

			cudaCheck(hipGraphicsUnmapResources(1, &imageResource, 0));
		}

		// draw fullscreen quad
		ShaderPtr s = Shader::shaders["fullscreen"];
		s->Use();
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, screenTexture);
		s->setInt("tex", 0);
		vao->Bind();
		glDrawArrays(GL_TRIANGLES, 0, 6);
		vao->Unbind();
		s->Unuse();
	}

	void CameraRaySnapshot()
	{
		delete lines; // ok if null (e.g. first instance)

		std::vector<glm::vec3> poss, dirs, tClrs, bClrs;

		glm::vec2 imgSize(screenDim);
		for (int x = 0; x < imgSize.x; x++)
		{
			for (int y = 0; y < imgSize.y; y++)
			{
				glm::vec2 screenCoord(
					(2.0f * x) / imgSize.x - 1.0f,
					(-2.0f * y) / imgSize.y + 1.0f);
				Ray ray = cam.makeRay(screenCoord);
				poss.push_back(ray.origin);
				dirs.push_back(ray.direction);
			}
		}

		for (int i = 0; i < poss.size(); i++)
		{
			tClrs.push_back(glm::vec3(1));
			bClrs.push_back(glm::vec3(0));
		}

		lines = new LinePool(poss, dirs, tClrs, bClrs);
	}
}
