#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "CommonDevice.cuh"
#include <stdio.h>
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
#include <math.h>

//#define N 25

#define MAX 100

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states)
{
	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		blockIdx.x * blockDim.x + threadIdx.x, /* the sequence number should be different for each core (unless you want all
									 cores to get the same sequence of numbers for some reason - use thread id! */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[blockIdx.x * blockDim.x + threadIdx.x]);
	//printf("%d\n", hiprand(&states[blockIdx.x * blockDim.x + threadIdx.x]));
}

__global__ void randoms(hiprandState_t* states)
{
	/* hiprand works like rand - except that it takes a state as a parameter */
	printf("%d\n", hiprand(&states[blockIdx.x * blockDim.x + threadIdx.x]) % 100);
}


void InitCUDARand(hiprandState_t*& states, unsigned N)
{
	printf("Allocating %d rand states.\n", N);

	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
		 we will store a random state for every thread  */
	//hiprandState_t* states;

	/* allocate space on the GPU for the random states */
	hipMalloc((void**)&states, N * sizeof(hiprandState_t));

	/* invoke the GPU to initialize all of the random states */
	init<<<N, 1>>>(time(0), states);
	hipDeviceSynchronize();

	// test
	//glm::vec2 screenDim = { 500, 265 };
	//const int KernelBlockSize = 256;
	//const int KernelNumBlocks = (screenDim.x * screenDim.y + KernelBlockSize - 1) / KernelBlockSize;
	//randoms<<<KernelNumBlocks, KernelBlockSize>>>(states);
	//hipDeviceSynchronize();
}

void ShutdownCUDARands(hiprandState_t*& states)
{
	hipFree(states);
}