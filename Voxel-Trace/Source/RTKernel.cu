#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "CommonDevice.cuh"
#include "Voxtrace.h"
#include "RayCamera.h"
#include "RTKernel.cuh"
#include "pick.h"

surface<void, 2> screenSurface;

__host__ surface<void, 2>& GetScreenSurface()
{
	return screenSurface;
}

struct PrimaryRayCaster
{
	__device__
	PrimaryRayCaster(ContextInfo& inf, glm::vec4& v) : info(inf), val(v) {}
	ContextInfo info;
	glm::vec4& val;
	int depthRemaining = 3;

	__device__
	bool operator()(
		glm::vec3 p, 
		Voxels::Block* block, 
		glm::vec3 norm, 
		glm::vec3 ex)
	{
		if (depthRemaining <= 0)
			return true;
		if (block)
		{
			if (block->alpha == 0)
				return false;

			// reflects, i am sorry
			bool refracted = false; // jank
			glm::vec3 refClr(block->diffuse);
			if (block->alpha < 1)
			{
				glm::vec3 reflDir = glm::normalize(glm::reflect(info.ray.direction, norm));
				PrimaryRayCaster castor = *this;
				castor.depthRemaining--;
				raycastBranchless(info.pWorld, info.worldDim, ex + reflDir * .001f, reflDir, 50.f, castor);
				return true; // uncomment when recursion is allowed
				//refClr = refrDir * .5f + .5f;
			}

			float visibility = 1;
			//int numShadowRays = numShadowRays;
			auto shadowCB = [&visibility, this](
				glm::vec3 p, Voxels::Block* block, glm::vec3 norm, glm::vec3)->bool
			{
				if (block && block->alpha == 1)
				{
					visibility -= 1.f / info.numShadowRays;
					return true;
				}
				return false;
			};

			glm::vec3 sunRay = glm::normalize(info.sun.position - ex); // block-to-light ray
			//raycastBranchless(pWorld, worldDim, ex + .02f * sunRay,
			//	sunRay, glm::min(glm::distance(sun.position, ex), 200.f), shadowCB);

			for (int i = 0; i < info.numShadowRays; i++)
			{
				float distToSun = glm::distance(info.sun.position, ex);
				float angle = glm::atan(info.sun.radius / distToSun);
				glm::vec3 shadowDir = RandVecInCone(info.sun.position - ex, angle, info.state);
				raycastBranchless(info.pWorld, info.worldDim, ex + .001f * shadowDir,
					shadowDir, glm::min(distToSun, 50.f), shadowCB);
				//block->diffuse = glm::vec3(angle * .5f + .5f);
			}

			//block->diffuse = shadowDir * .5f + .5f;
			//block->diffuse = ex / 2.f;

			// phong
			float diff = glm::max(glm::dot(sunRay, norm), 0.f);
			float spec = glm::pow(glm::max(glm::dot(info.ray.direction,
				glm::reflect(sunRay, norm)), 0.0f), 64.f);
			glm::vec3 ambient = glm::vec3(.2) * block->diffuse;
			glm::vec3 specular = glm::vec3(.7) * spec;
			glm::vec3 diffuse = block->diffuse * diff;

			diffuse *= visibility;
			specular *= visibility;

			// final color of pixel
			glm::vec3 FragColor(0);
			FragColor = diffuse + ambient + specular;
			val = glm::vec4(FragColor, 1.f);
			return true;
		}
		return false;
	};
};



__global__ void epicRayTracer(Voxels::Block* pWorld, glm::ivec3 worldDim,
	PerspectiveRayCamera camera, int numShadowRays, glm::vec2 imgSize,
	glm::vec3 chunkDim, Voxels::Light sun, hiprandState_t* states)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int n = imgSize.x * imgSize.y;

	//printf("index = %d, stride = %d, n = %d\n", index, stride, n);
	for (int i = index; i < n; i += stride)
	{
		glm::vec2 imgPos = expand(i, imgSize.y);
		//glm::vec2 imgPos(x, y);
		glm::vec2 screenCoord(
			(2.0f * imgPos.x) / imgSize.x - 1.0f,
			(-2.0f * imgPos.y) / imgSize.y + 1.0f);
		Ray ray = camera.makeRay(screenCoord);


		// TODO: move all this into its own function so it can call itself recursively, etc

		glm::vec4 val{ .53f, .81f, .92f, 1 };
		PrimaryRayCaster primRay(
			ContextInfo(pWorld, worldDim, numShadowRays, sun, states[index], ray), val);
		raycastBranchless(pWorld, worldDim, ray.origin, ray.direction, 50, primRay);

		// write final pixel value
		surf2Dwrite(val, screenSurface, imgPos.x * sizeof(val), imgSize.y - imgPos.y - 1);
	}
}